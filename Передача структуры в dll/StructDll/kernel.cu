#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cmath>
#include <ctime>
#include <iostream>
#include <chrono>
#include <limits>
#define pointsCount 8
#define n 13


typedef struct
{
	double a, k;
} Input;

typedef struct
{
	double y, x;
} Output;

extern "C" __declspec(dllexport) void   withStruct(Input cpuInArray[], Output cpuOutArray[], int arraySize);
__global__ void sampleStructFunction(Output* op, Input* ip, int size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < size) {
		double x0;
		double minc;
		double minx = 0;
		double maxx = 1;
		double besty = 0;
		float bestx = 0;

		double step;
		double minx0;
		for (int j = 1; j <= n; j++)
		{
			step = (maxx - minx) / pointsCount;
			x0 = (double)(step + minx) / 2;
			minc = ip[idx].a * x0 * x0 * x0 + ip[idx].k;
			minx0 = x0;

			for (int i = 1; i < pointsCount; i++)
			{
				x0 = (double)(i + 1) * step + minx;
				double m = ip[idx].a * x0 * x0 * x0 + ip[idx].k;
				if (fabsf(minc) > fabsf(m))
				{
					minc = m;
					minx0 = x0;
				}
			}

			minx = minx0 - step; maxx = minx0 + step;


			besty = minc;
			bestx = minx0;

		}
		op[idx].y = besty;
		op[idx].x = bestx;
	}
}


int main()
{
/*	int arraySize = 512 * 50000;
	int InSize = sizeof(Input);
	int numBytesIn = arraySize * InSize;
	Input* cpuInArray;
	hipMalloc((void**)& cpuInArray, numBytesIn);


	int OutSize = sizeof(Output);
	int numBytesOut = arraySize * OutSize;
	Output* cpuOutArray;
	hipMalloc((void**)& cpuOutArray, numBytesOut);
	srand(time(0));
	for (int i = 0; i < arraySize; i++)
	{
		cpuInArray[i].a = rand() % 10 + rand() / double(RAND_MAX) - 5;
		cpuInArray[i].k = rand() % 10 + rand() / double(RAND_MAX) - 5;
		cpuOutArray[i].y = 0;
		cpuOutArray[i].x = 0;
	}

	 withStruct(cpuInArray, cpuOutArray, arraySize);*/
	
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
extern "C" __declspec(dllexport) void withStruct(Input cpuInArray[], Output cpuOutArray[], int arraySize)
{
	float timerValueGPU, timerValueCPU;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	int InSize = sizeof(Input);
	int numBytesIn = arraySize * InSize;
	Input * gpuIntArray;
	hipMalloc((void**)& gpuIntArray, numBytesIn);


	int OutSize = sizeof(Output);
	int numBytesOut = arraySize * OutSize;
	Output* gpuOutArray;
	hipMalloc((void**)& gpuOutArray, numBytesOut);

	hipError_t cudaStatus;
	int N_thread = 512; // ����� ����� � �����
	int N_blocks = (int)arraySize / N_thread;
	if ((arraySize % N_thread) == 0)
	{
		N_blocks = arraySize / N_thread;
	}
	else
	{
		N_blocks = (int)(arraySize / N_thread) + 1;
	}
	dim3 blocks(N_blocks);

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	hipEventRecord(start, 0);
	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(gpuIntArray, cpuInArray, arraySize * sizeof(Input), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.

	printf("begin\r\n");
	sampleStructFunction << <N_blocks, N_thread >> > (gpuOutArray, gpuIntArray, arraySize);
	printf("the end sampleFunction\r\n");
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(cpuOutArray, gpuOutArray, arraySize * sizeof(Output), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&timerValueGPU, start, stop);
	printf("\n GPU calculation time: %f ms\n", timerValueGPU);
Error:
	hipFree(gpuIntArray);
	hipFree(gpuOutArray);
}
